#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm_batched<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float** A, const float** B, const float beta,
    float** C, int batch_size) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemmBatched(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N, batch_size));
}

template <>
void caffe_gpu_gemm_batched<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double** A, const double** B, const double beta,
    double** C, int batch_size) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemmBatched(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N, batch_size));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}



template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}


template <typename Dtype>
__global__ void add_elementwise_kernel(const int n, const Dtype* x, Dtype* y, const int m) {
  CUDA_KERNEL_LOOP(index, n) {
	  Dtype acum = 0;
	  for (int i = index; i < m + index; i++)
		  acum += x[i];
	  y[index] = acum;
  }
}


template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}


template <>
void caffe_gpu_add_elementwise<float>(const int N, const float* x, float* y, const int M) {
  // NOLINT_NEXT_LINE(whitespace/operators)
	add_elementwise_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, x, y, M );
}

template <>
void caffe_gpu_add_elementwise<double>(const int N, const double* x, double* y, const int M) {
  // NOLINT_NEXT_LINE(whitespace/operators)
	add_elementwise_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
	  N, x, y, M);
}


template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}
#include <stdio.h>

template <typename Dtype>
__global__ void mul_kernel_batched(const int n, const Dtype* a,
    const Dtype* b, Dtype* y, const int m) {
	unsigned int batch_offset = 0;
 {
   CUDA_KERNEL_LOOP(index, m) {
	  Dtype b_at_index = b[index];
	  for (unsigned int batch_offset = 0; batch_offset < n; batch_offset+=m)
		y[index + batch_offset] = a[index + batch_offset] * b_at_index;
  }
 }
}


template <typename Dtype>
__global__ void mul_kernel_batched_v2(const int n, const Dtype* a,
    const Dtype* b, Dtype* y, const int m) {
	unsigned int batch_offset = 0;

 {
  CUDA_KERNEL_LOOP(index, m) {
	  for (unsigned int batch_offset = 0; batch_offset < n; batch_offset+=m)
		y[index + batch_offset] = a[index + batch_offset] * b[index];
  }
 }
}


template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y, const int M) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  if (M <= 0 || M > N)
	  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y);
  else
	  mul_kernel_batched<float><<<CAFFE_GET_BLOCKS(M), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y, M);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y, const int M) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  if (M <= 0 || M > N)
	  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y);
  else
	  mul_kernel_batched<double><<<CAFFE_GET_BLOCKS(M), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y, M);
}


__global__ void clip_lower_kernel_double(const int n, const double lower_bound, const double* x, double* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = fmax(x[index], lower_bound);
  }
}

__global__ void clip_lower_kernel_float(const int n, const float lower_bound, const float* x, float* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = fmaxf(x[index], lower_bound);
  }
}

template <>
void caffe_gpu_clip_lower<float>(const int N, const float lower_bound, const float* x, float* y) {
	clip_lower_kernel_float<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, lower_bound, x, y);
}

template <>
void caffe_gpu_clip_lower<double>(const int N, const double lower_bound, const double* x, double* y) {
	clip_lower_kernel_double<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, lower_bound, x, y);
}


__global__ void clip_upper_kernel_double(const int n, const double lower_bound, const double* x, double* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = fmin(x[index], lower_bound);
  }
}

__global__ void clip_upper_kernel_float(const int n, const float lower_bound, const float* x, float* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = fminf(x[index], lower_bound);
  }
}

template <>
void caffe_gpu_clip_upper<float>(const int N, const float lower_bound, const float* x, float* y) {
	clip_upper_kernel_float<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, lower_bound, x, y);
}

template <>
void caffe_gpu_clip_upper<double>(const int N, const double lower_bound, const double* x, double* y) {
	clip_upper_kernel_double<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, lower_bound, x, y);
}



template <typename Dtype>
void caffe_gpu_clip_upper(const int N, const Dtype upper_bound, const Dtype* x, Dtype* y) {

}


template <typename Dtype>
__global__ void clip_eps_kernel(const int n, const Dtype eps_bound, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
	Dtype val = x[index];
    y[index] = abs(val) > eps_bound ? val : 0;
  }
}


template <>
void caffe_gpu_clip_eps<float>(const int N, const float eps_bound, const float* x, float* y) {
	clip_eps_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, eps_bound, x, y);
}
template <>
void caffe_gpu_clip_eps<double>(const int N, const double eps_bound, const double* x, double* y) {
	clip_eps_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, eps_bound, x, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

__global__ void popc_kernel(const int n, const float* a,
    const float* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                      static_cast<uint32_t>(b[index]));
  }
}

__global__ void popcll_kernel(const int n, const double* a,
    const double* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                      static_cast<uint64_t>(b[index]));
  }
}

template <>
uint32_t caffe_gpu_hamming_distance<float>(const int n, const float* x,
                                  const float* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double>(const int n, const double* x,
                                   const double* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}


#include <cub/hipcub/hipcub.hpp>

using namespace hipcub;

/**
 * Segmented reduction that uses d_out values as intialization values (one block per segment)
 */
template <
    typename                ChainedPolicyT,             ///< Chained tuning policy
    typename                InputIteratorT,             ///< Random-access input iterator type for reading input items \iterator
    typename                OutputIteratorT,            ///< Output iterator type for recording the reduced aggregate \iterator
    typename                OffsetT,                    ///< Signed integer type for global offsets
    typename                ReductionOpT,               ///< Binary reduction functor type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename                T>                          ///< Data element type that is convertible to the \p value type of \p InputIteratorT
__launch_bounds__ (int(ChainedPolicyT::ActivePolicy::ReducePolicy::BLOCK_THREADS))
__global__ void MyDeviceSegmentedReduceWithInitKernel(
    InputIteratorT          d_in,                       ///< [in] Pointer to the input sequence of data items
    OutputIteratorT         d_out,                      ///< [out] Pointer to the output aggregate
    int                     *d_begin_offsets,           ///< [in] %Devic-accessible pointer to the sequence of beginning offsets of length \p num_segments, such that <tt>d_begin_offsets[i]</tt> is the first element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>
    int                     *d_end_offsets,             ///< [in] %Device-accessible pointer to the sequence of ending offsets of length \p num_segments, such that <tt>d_end_offsets[i]-1</tt> is the last element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>.  If <tt>d_end_offsets[i]-1</tt> <= <tt>d_begin_offsets[i]</tt>, the <em>i</em><sup>th</sup> is considered empty.
    int                     num_segments,               ///< [in] The number of segments that comprise the sorting data
    ReductionOpT            reduction_op)               ///< [in] Binary reduction functor

{
    // Thread block type for reducing input tiles
    typedef AgentReduce<
            typename ChainedPolicyT::ActivePolicy::ReducePolicy,
            InputIteratorT,
            OffsetT,
            ReductionOpT>
        AgentReduceT;

    // Shared memory storage
    __shared__ typename AgentReduceT::TempStorage temp_storage;

    OffsetT segment_begin   = d_begin_offsets[blockIdx.x];
    OffsetT segment_end     = d_end_offsets[blockIdx.x];

    // Check if empty problem
    if (segment_begin == segment_end)
    {
        return;
    }

    // Consume input tiles
    T block_aggregate = AgentReduceT(temp_storage, d_in, reduction_op).ConsumeRange(
        segment_begin,
        segment_end);

    // Normalize as needed
    NormalizeReductionOutput(block_aggregate, segment_begin, d_in);

    if (threadIdx.x == 0)
        d_out[blockIdx.x] = reduction_op(d_out[blockIdx.x], block_aggregate);
    	//d_out[blockIdx.x] = reduction_op((T)0, block_aggregate);
}

/**
 * Utility class for dispatching the appropriately-tuned kernels for device-wide reduction
 */
template <
    typename InputIteratorT,    ///< Random-access input iterator type for reading input items \iterator
    typename OutputIteratorT,   ///< Output iterator type for recording the reduced aggregate \iterator
    typename OffsetT,           ///< Signed integer type for global offsets
    typename ReductionOpT>      ///< Binary reduction functor type having member <tt>T operator()(const T &a, const T &b)</tt>
struct MyDispatchSegmentedReduce :
    DeviceReducePolicy<
        typename std::iterator_traits<InputIteratorT>::value_type,
        OffsetT,
        ReductionOpT>
{
    //------------------------------------------------------------------------------
    // Constants
    //------------------------------------------------------------------------------

    // Data type of input iterator
    typedef typename std::iterator_traits<InputIteratorT>::value_type T;


    //------------------------------------------------------------------------------
    // Problem state
    //------------------------------------------------------------------------------

    void                *d_temp_storage;        ///< [in] %Device-accessible allocation of temporary storage.  When NULL, the required allocation size is written to \p temp_storage_bytes and no work is done.
    size_t              &temp_storage_bytes;    ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation
    InputIteratorT      d_in;                   ///< [in] Pointer to the input sequence of data items
    OutputIteratorT     d_out;                  ///< [out] Pointer to the output aggregate
    OffsetT             num_segments;           ///< [in] The number of segments that comprise the sorting data
    OffsetT             *d_begin_offsets;       ///< [in] %Device-accessible pointer to the sequence of beginning offsets of length \p num_segments, such that <tt>d_begin_offsets[i]</tt> is the first element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>
    OffsetT             *d_end_offsets;         ///< [in] %Device-accessible pointer to the sequence of ending offsets of length \p num_segments, such that <tt>d_end_offsets[i]-1</tt> is the last element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>.  If <tt>d_end_offsets[i]-1</tt> <= <tt>d_begin_offsets[i]</tt>, the <em>i</em><sup>th</sup> is considered empty.
    ReductionOpT        reduction_op;           ///< [in] Binary reduction functor
    hipStream_t        stream;                 ///< [in] CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
    bool                debug_synchronous;      ///< [in] Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    int                 ptx_version;            ///< [in] PTX version

    //------------------------------------------------------------------------------
    // Constructor
    //------------------------------------------------------------------------------

    /// Constructor
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    MyDispatchSegmentedReduce(
        void*                   d_temp_storage,
        size_t                  &temp_storage_bytes,
        InputIteratorT          d_in,
        OutputIteratorT         d_out,
        OffsetT                 num_segments,
        OffsetT                 *d_begin_offsets,
        OffsetT                 *d_end_offsets,
        ReductionOpT            reduction_op,
        hipStream_t            stream,
        bool                    debug_synchronous,
        int                     ptx_version)
    :
        d_temp_storage(d_temp_storage),
        temp_storage_bytes(temp_storage_bytes),
        d_in(d_in),
        d_out(d_out),
        num_segments(num_segments),
        d_begin_offsets(d_begin_offsets),
        d_end_offsets(d_end_offsets),
        reduction_op(reduction_op),
        stream(stream),
        debug_synchronous(debug_synchronous),
        ptx_version(ptx_version)
    {}



    //------------------------------------------------------------------------------
    // Chained policy invocation
    //------------------------------------------------------------------------------

    /// Invocation
    template <
        typename                        ActivePolicyT,                  ///< Umbrella policy active for the target device
        typename                        DeviceSegmentedReduceKernelT>   ///< Function type of hipcub::DeviceSegmentedReduceKernel
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    hipError_t InvokePasses(
        DeviceSegmentedReduceKernelT    segmented_reduce_kernel)        ///< [in] Kernel function pointer to parameterization of hipcub::DeviceSegmentedReduceKernel
    {
#ifndef CUB_RUNTIME_ENABLED

        // Kernel launch not supported from this device
        return HipcubDebug(hipErrorNotSupported );
#else
        hipError_t error = hipSuccess;
        do
        {
            // Return if the caller is simply requesting the size of the storage allocation
            if (d_temp_storage == NULL)
            {
                temp_storage_bytes = 1;
                return hipSuccess;
            }

            // Init kernel configuration
            KernelConfig segmented_reduce_config;
            if (HipcubDebug(error = segmented_reduce_config.Init<typename ActivePolicyT::SegmentedReducePolicy>(segmented_reduce_kernel))) break;

            // Log device_reduce_sweep_kernel configuration
            if (debug_synchronous) _HipcubLog("Invoking MySegmentedDeviceReduceKernel<<<%d, %d, 0, %lld>>>(), %d items per thread, %d SM occupancy\n",
                num_segments,
                ActivePolicyT::SegmentedReducePolicy::BLOCK_THREADS,
                (long long) stream,
                ActivePolicyT::SegmentedReducePolicy::ITEMS_PER_THREAD,
                segmented_reduce_config.sm_occupancy);

            // Invoke DeviceReduceKernel
            segmented_reduce_kernel<<<num_segments, ActivePolicyT::SegmentedReducePolicy::BLOCK_THREADS, 0, stream>>>(
                d_in,
                d_out,
                d_begin_offsets,
                d_end_offsets,
                num_segments,
                reduction_op);

            // Check for failure to launch
            if (HipcubDebug(error = hipPeekAtLastError())) break;

            // Sync the stream if specified to flush runtime errors
            if (debug_synchronous && (HipcubDebug(error = SyncStream(stream)))) break;
        }
        while (0);

        return error;

#endif // CUB_RUNTIME_ENABLED

    }


    /// Invocation
    template <typename ActivePolicyT>
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    hipError_t Invoke()
    {
        typedef typename MyDispatchSegmentedReduce::MaxPolicy MaxPolicyT;

        // Force kernel code-generation in all compiler passes
        return InvokePasses<ActivePolicyT>(
        	MyDeviceSegmentedReduceWithInitKernel<MaxPolicyT, InputIteratorT, OutputIteratorT, OffsetT, ReductionOpT, T>);
    }


    //------------------------------------------------------------------------------
    // Dispatch entrypoints
    //------------------------------------------------------------------------------

    /**
     * Internal dispatch routine for computing a device-wide reduction
     */
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    static hipError_t Dispatch(
        void            *d_temp_storage,                    ///< [in] %Device-accessible allocation of temporary storage.  When NULL, the required allocation size is written to \p temp_storage_bytes and no work is done.
        size_t          &temp_storage_bytes,                ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation
        InputIteratorT  d_in,                               ///< [in] Pointer to the input sequence of data items
        OutputIteratorT d_out,                              ///< [out] Pointer to the output aggregate
        int             num_segments,                       ///< [in] The number of segments that comprise the sorting data
        int             *d_begin_offsets,                   ///< [in] %Device-accessible pointer to the sequence of beginning offsets of length \p num_segments, such that <tt>d_begin_offsets[i]</tt> is the first element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>
        int             *d_end_offsets,                     ///< [in] %Device-accessible pointer to the sequence of ending offsets of length \p num_segments, such that <tt>d_end_offsets[i]-1</tt> is the last element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>.  If <tt>d_end_offsets[i]-1</tt> <= <tt>d_begin_offsets[i]</tt>, the <em>i</em><sup>th</sup> is considered empty.
        ReductionOpT    reduction_op,                       ///< [in] Binary reduction functor
        hipStream_t    stream,                             ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool            debug_synchronous)                  ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        typedef typename MyDispatchSegmentedReduce::MaxPolicy MaxPolicyT;

        if (num_segments <= 0)
            return hipSuccess;

        hipError_t error = hipSuccess;
        do
        {
            // Get PTX version
            int ptx_version;
            if (HipcubDebug(error = PtxVersion(ptx_version))) break;

            // Create dispatch functor
            MyDispatchSegmentedReduce dispatch(
                d_temp_storage, temp_storage_bytes,
                d_in, d_out,
                num_segments, d_begin_offsets, d_end_offsets,
                reduction_op,
                stream, debug_synchronous, ptx_version);

            // Dispatch to chained policy
            if (HipcubDebug(error = MaxPolicyT::Invoke(ptx_version, dispatch))) break;
        }
        while (0);

        return error;
    }
};

template <
	typename            InputIteratorT,
	typename            OutputIteratorT>
HIPCUB_RUNTIME_FUNCTION
static hipError_t segmentedSumWithAdd(
	void                *d_temp_storage,                    ///< [in] %Device-accessible allocation of temporary storage.  When NULL, the required allocation size is written to \p temp_storage_bytes and no work is done.
	size_t              &temp_storage_bytes,                ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation
	InputIteratorT      d_in,                               ///< [in] Pointer to the input sequence of data items
	OutputIteratorT     d_out,                              ///< [out] Pointer to the output aggregate
	int                 num_segments,                       ///< [in] The number of segments that comprise the sorting data
	int                 *d_begin_offsets,                   ///< [in] %Device-accessible pointer to the sequence of beginning offsets of length \p num_segments, such that <tt>d_begin_offsets[i]</tt> is the first element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>
	int                 *d_end_offsets,                     ///< [in] %Device-accessible pointer to the sequence of ending offsets of length \p num_segments, such that <tt>d_end_offsets[i]-1</tt> is the last element of the <em>i</em><sup>th</sup> data segment in <tt>d_keys_*</tt> and <tt>d_values_*</tt>.  If <tt>d_end_offsets[i]-1</tt> <= <tt>d_begin_offsets[i]</tt>, the <em>i</em><sup>th</sup> is considered empty.
	hipStream_t        stream              = 0,            ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
	bool                debug_synchronous   = false)        ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
{
	typedef int OffsetT;                                                    // Signed integer type for global offsets
	typedef typename std::iterator_traits<InputIteratorT>::value_type T;    // Data element type

	return MyDispatchSegmentedReduce<InputIteratorT, OutputIteratorT, OffsetT, hipcub::Sum>::Dispatch(
		d_temp_storage,
		temp_storage_bytes,
		d_in,
		d_out,
		num_segments,
		d_begin_offsets,
		d_end_offsets,
		hipcub::Sum(),
		stream,
		debug_synchronous);
}



template <typename Dtype>
void caffe_gpu_sum(const int n, const Dtype* x, Dtype* y, const int m) {
	CHECK_EQ(n % m, 0);
	int num_segments = n/m;

	int* offsets = new int[num_segments + 1];

	offsets[0] = 0;

	for (int i = 0; i < num_segments; i++) offsets[i+1] = m*(i+1);

	int* offsets_d;
	CUDA_CHECK(hipMalloc(&offsets_d, sizeof(int)*(num_segments+1)));

	caffe_gpu_memcpy(sizeof(int)*(num_segments + 1), offsets, offsets_d);

	caffe_gpu_sum(n, x, y, num_segments, offsets_d , (hipStream_t)NULL);

	delete offsets;
}



template <typename Dtype>
void caffe_gpu_sum(const int n, const Dtype* x, Dtype* y, const int num_segments, int* offsets_gpu, hipStream_t streamId) {

	// DeviceSegmentedReduce in version 1.5.1 always returns temp_storage_bytes=1 and never actually uses allocated storage
	// so we can just use non-zero value for temp storage and avoid getting temp_storage_bytes size
	size_t temp_storage_bytes = 0;
	void* temp_storage_d = (void*)1;

	//CUDA_CHECK(hipcub::DeviceSegmentedReduce::Sum(NULL, temp_storage_bytes, x, y,  num_segments, offsets_d, offsets_d + 1, streamId));
	//CUDA_CHECK(hipMalloc(&temp_storage_d, temp_storage_bytes));

//	CUDA_CHECK(hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes, x, y,  1024, streamId));
//	CUDA_CHECK(hipMalloc(&temp_storage_d, temp_storage_bytes));

//	CUDA_CHECK(hipcub::DeviceReduce::Sum(temp_storage_d, temp_storage_bytes, x, y,  1024, streamId, false));

	//CUDA_CHECK(hipcub::DeviceSegmentedReduce::Sum(temp_storage_d, temp_storage_bytes, x, y,  num_segments, offsets_gpu, offsets_gpu + 1, streamId, false));
	CUDA_CHECK(segmentedSumWithAdd(temp_storage_d, temp_storage_bytes, x, y,  num_segments, offsets_gpu, offsets_gpu + 1, streamId, false));

//	CUDA_CHECK(hipFree(temp_storage_d));
}



template void caffe_gpu_sum<float>(const int n, const float* x, float* y, const int num_segments, int* offsets_gpu, hipStream_t streamId);
template void caffe_gpu_sum<double>(const int n, const double* x, double* y, const int num_segments, int* offsets_gpu, hipStream_t streamId);

template void caffe_gpu_sum<float>(const int n, const float* x, float* y, const int m);
template void caffe_gpu_sum<double>(const int n, const double* x, double* y, const int m);


template <typename Dtype>
void caffe_cpu_sum(const int n, const Dtype* x, Dtype* y, int m) {
	if (m <= 0)
		m = n;
	int num_segments = n/m;

	int input_offset = 0;
	for (int i = 0; i < num_segments; ++i) {

		Dtype sum_value = 0;

		for (int j = 0; j < m; ++j) {
			sum_value += x[input_offset];
			input_offset++;
		}

		y[i] += sum_value;
	}
}

template void caffe_cpu_sum(const int n, const double* x, double* y, int m);
template void caffe_cpu_sum(const int n, const float* x, float* y, int m);

}  // namespace caffe
